#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 10


__global__ void add(int *a,int *b,int *c ) {

	int tid = blockIdx.x;
	printf("Executing on %d\n",tid);
	if(tid<N){
		c[tid] = a[tid] + b[tid];
	}
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void) {

	int a[N],b[N],c[N];
	int *ad,*bd,*cd;

	hipMalloc((int **)&ad,N*sizeof(int));
	hipMalloc((int **)&bd,N*sizeof(int));
	hipMalloc((int **)&cd,N*sizeof(int));

	for (int i=0; i<N; i++) {
		a[i] = i * 2;
		b[i] = i * 3;
	}

	printf("[");
	for (int i=0; i<N; i++) {
		printf("%d,",a[i]);
	}
	printf("]\n");
	printf("[");
	for (int i=0; i<N; i++) {
		printf("%d,",b[i]);
	}
	printf("]\n");

	hipMemcpy(ad,&a,N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(bd,&b,N*sizeof(int),hipMemcpyHostToDevice);

	add<<<N,1>>>(ad,bd,cd);

	hipMemcpy(&c,cd,N*sizeof(int),hipMemcpyDeviceToHost);

	printf("Addition:");
	printf("[");
	for (int i=0; i<N; i++) {
			printf("%d,",c[i]);
		}
	printf("]");

	hipFree(ad);
	hipFree(bd);
	hipFree(cd);

	return 0;
}
